#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <stdio.h>

__global__ void hello() {
    printf("Hello!\n");
}

int main() {
    
    hipProfilerStart();
    hello<<<1, 1>>>();
    fflush(stdout);
    hipProfilerStop();
    return 0;
}